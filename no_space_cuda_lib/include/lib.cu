#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "lib.h"



namespace detail {
__global__ void lib_kernel()
{
    printf("\n I am thread %d from lib_kernel\n", threadIdx.x);
}
}  // namespace detail

void Lib::run()
{
    detail::lib_kernel<<<1, 1>>>();   
}