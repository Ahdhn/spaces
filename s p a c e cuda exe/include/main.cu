#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void exec_kernel()
{
    printf("\n I am thread %d from exec_kernel\n", threadIdx.x);
}

int main(int argc, char** argv)
{
    exec_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
}